#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <hip/hip_runtime.h>
#define BYTE unsigned char

using namespace std;

class aes_block
{
public:
    BYTE block[16];
};

void printBytes(BYTE b[], int len) {
int i;
for (i=0; i<len; i++)
    printf("%x ", b[i]);
printf("\n");
}


void fileOnePrintBytes(BYTE b[], int len, FILE* fp) {
int i;
for (i=0; i<len; i++)
   fprintf(fp, "%02x ", b[i]);
fprintf(fp, "\n");
}


int flag=0;
void fileTwoPrintBytes(BYTE b[], int len, FILE* fp) {
int i;
for (i=0; i<len; i++){
   fprintf(fp, "%c", b[i]);
   if(b[i]=='\n')
        flag++;
   }

}


void fileThreePrintBytes(BYTE b[], int len, FILE* fp) {
int i;
for (i=0; i<len; i++){
 if(b[i]=='\0'){
   return ;
   }
   fprintf(fp, "%c", b[i]);
   if(b[i]=='\n')
        flag++;
   }
}


BYTE SBox[] =
{   /*0    1    2    3    4    5    6    7    8    9    a    b    c    d    e    f */
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76, /*0*/ 
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0, /*1*/
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15, /*2*/
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75, /*3*/
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84, /*4*/
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf, /*5*/
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8, /*6*/ 
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2, /*7*/
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73, /*8*/
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb, /*9*/
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79, /*a*/
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08, /*b*/
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a, /*c*/
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e, /*d*/
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf, /*e*/
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16  /*f*/
};


__device__ void AES_SubBytes(BYTE state[], BYTE sbox[]) {
int i;
for(i = 0; i < 16; i++)
    state[i] = sbox[state[i]];
}

__device__ void AES_AddRoundKey(BYTE state[], BYTE rkey[]) {
    int i;
    for(i = 0; i < 16; i++)
        state[i] ^= rkey[i];
}

__device__ void AES_ShiftRows(BYTE state[], BYTE shifttab[]) {
    BYTE h[16];
    memcpy(h, state, 16);
    int i;
    for(i = 0; i < 16; i++)
        state[i] = h[shifttab[i]];
}

__device__ void AES_MixColumns(BYTE state[], BYTE AES_xtime[]) {
    int i;
    //ared__ BYTE AES_xtime[];
for(i = 0; i < 16; i += 4) {
    BYTE s0 = state[i + 0], s1 = state[i + 1];
    BYTE s2 = state[i + 2], s3 = state[i + 3];
    BYTE h = s0 ^ s1 ^ s2 ^ s3;
    state[i + 0] ^= h ^ AES_xtime[s0 ^ s1];
    state[i + 1] ^= h ^ AES_xtime[s1 ^ s2];
    state[i + 2] ^= h ^ AES_xtime[s2 ^ s3];
    state[i + 3] ^= h ^ AES_xtime[s3 ^ s0];
}
}

__device__ void AES_MixColumns_Inv(BYTE state[], BYTE AES_xtime[]) {
    int i;
    for(i = 0; i < 16; i += 4) {
        BYTE s0 = state[i + 0], s1 = state[i + 1];
        BYTE s2 = state[i + 2], s3 = state[i + 3];
        BYTE h = s0 ^ s1 ^ s2 ^ s3;
        BYTE xh = AES_xtime[h];
        BYTE h1 = AES_xtime[AES_xtime[xh ^ s0 ^ s2]] ^ h;
        BYTE h2 = AES_xtime[AES_xtime[xh ^ s1 ^ s3]] ^ h;
        state[i + 0] ^= h1 ^ AES_xtime[s0 ^ s1];
        state[i + 1] ^= h2 ^ AES_xtime[s1 ^ s2];
        state[i + 2] ^= h1 ^ AES_xtime[s2 ^ s3];
        state[i + 3] ^= h2 ^ AES_xtime[s3 ^ s0];
}
}

__device__ void AES_Init(BYTE SBox[], BYTE AES_ShiftRowTab[], BYTE SBox_Inv[], BYTE AES_xtime[], BYTE AES_ShiftRowTab_Inv[]) {
    // BYTE AES_ShiftRowTab[16] ;
    AES_ShiftRow[0]=0;
    AES_ShiftRow[1]=5;
    AES_ShiftRow[2]=10;
    AES_ShiftRow[3]=15;
    AES_ShiftRow[4]=4;
    AES_ShiftRow[5]=9;
    AES_ShiftRow[6]=14;
    AES_ShiftRow[7]=3;
    AES_ShiftRow[8]=8;
    AES_ShiftRow[9]=13;
    AES_ShiftRow[10]=2;
    AES_ShiftRow[11]=7;
    AES_ShiftRow[12]=12;
    AES_ShiftRow[13]=1;
    AES_ShiftRow[14]=6;
    AES_ShiftRow[15]=11;
    
SBox[0] = 0x63;SBox[1] = 0x7c;SBox[2] = 0x77;SBox[3] = 0x7b;SBox[4] = 0xf2;SBox[5] = 0x6b;SBox[6] = 0x6f;SBox[7] = 0xc5;SBox[8] = 0x30;SBox[9] = 0x1;SBox[10] = 0x67;SBox[11] = 0x2b;SBox[12] = 0xfe;SBox[13] = 0xd7;SBox[14] = 0xab;SBox[15] = 0x76;
SBox[16] = 0xca;SBox[17] = 0x82;SBox[18] = 0xc9;SBox[19] = 0x7d;SBox[20] = 0xfa;SBox[21] = 0x59;SBox[22] = 0x47;SBox[23] = 0xf0;SBox[24] = 0xad;SBox[25] = 0xd4;SBox[26] = 0xa2;SBox[27] = 0xaf;SBox[28] = 0x9c;SBox[29] = 0xa4;SBox[30] = 0x72;SBox[31] = 0xc0;
SBox[32] = 0xb7;SBox[33] = 0xfd;SBox[34] = 0x93;SBox[35] = 0x26;SBox[36] = 0x36;SBox[37] = 0x3f;SBox[38] = 0xf7;SBox[39] = 0xcc;SBox[40] = 0x34;SBox[41] = 0xa5;SBox[42] = 0xe5;SBox[43] = 0xf1;SBox[44] = 0x71;SBox[45] = 0xd8;SBox[46] = 0x31;SBox[47] = 0x15;
SBox[48] = 0x4;SBox[49] = 0xc7;SBox[50] = 0x23;SBox[51] = 0xc3;SBox[52] = 0x18;SBox[53] = 0x96;SBox[54] = 0x5;SBox[55] = 0x9a;SBox[56] = 0x7;SBox[57] = 0x12;SBox[58] = 0x80;SBox[59] = 0xe2;SBox[60] = 0xeb;SBox[61] = 0x27;SBox[62] = 0xb2;SBox[63] = 0x75;
SBox[64] = 0x9;SBox[65] = 0x83;SBox[66] = 0x2c;SBox[67] = 0x1a;SBox[68] = 0x1b;SBox[69] = 0x6e;SBox[70] = 0x5a;SBox[71] = 0xa0;SBox[72] = 0x52;SBox[73] = 0x3b;SBox[74] = 0xd6;SBox[75] = 0xb3;SBox[76] = 0x29;SBox[77] = 0xe3;SBox[78] = 0x2f;SBox[79] = 0x84;
SBox[80] = 0x53;SBox[81] = 0xd1;SBox[82] = 0x0;SBox[83] = 0xed;SBox[84] = 0x20;SBox[85] = 0xfc;SBox[86] = 0xb1;SBox[87] = 0x5b;SBox[88] = 0x6a;SBox[89] = 0xcb;SBox[90] = 0xbe;SBox[91] = 0x39;SBox[92] = 0x4a;SBox[93] = 0x4c;SBox[94] = 0x58;SBox[95] = 0xcf;
SBox[96] = 0xd0;SBox[97] = 0xef;SBox[98] = 0xaa;SBox[99] = 0xfb;SBox[100] = 0x43;SBox[101] = 0x4d;SBox[102] = 0x33;SBox[103] = 0x85;SBox[104] = 0x45;SBox[105] = 0xf9;SBox[106] = 0x2;SBox[107] = 0x7f;SBox[108] = 0x50;SBox[109] = 0x3c;SBox[110] = 0x9f;SBox[111] = 0xa8;
SBox[112] = 0x51;SBox[113] = 0xa3;SBox[114] = 0x40;SBox[115] = 0x8f;SBox[116] = 0x92;SBox[117] = 0x9d;SBox[118] = 0x38;SBox[119] = 0xf5;SBox[120] = 0xbc;SBox[121] = 0xb6;SBox[122] = 0xda;SBox[123] = 0x21;SBox[124] = 0x10;SBox[125] = 0xff;SBox[126] = 0xf3;SBox[127] = 0xd2;
SBox[128] = 0xcd;SBox[129] = 0xc;SBox[130] = 0x13;SBox[131] = 0xec;SBox[132] = 0x5f;SBox[133] = 0x97;SBox[134] = 0x44;SBox[135] = 0x17;SBox[136] = 0xc4;SBox[137] = 0xa7;SBox[138] = 0x7e;SBox[139] = 0x3d;SBox[140] = 0x64;SBox[141] = 0x5d;SBox[142] = 0x19;SBox[143] = 0x73;
SBox[144] = 0x60;SBox[145] = 0x81;SBox[146] = 0x4f;SBox[147] = 0xdc;SBox[148] = 0x22;SBox[149] = 0x2a;SBox[150] = 0x90;SBox[151] = 0x88;SBox[152] = 0x46;SBox[153] = 0xee;SBox[154] = 0xb8;SBox[155] = 0x14;SBox[156] = 0xde;SBox[157] = 0x5e;SBox[158] = 0xb;SBox[159] = 0xdb;
SBox[160] = 0xe0;SBox[161] = 0x32;SBox[162] = 0x3a;SBox[163] = 0xa;SBox[164] = 0x49;SBox[165] = 0x6;SBox[166] = 0x24;SBox[167] = 0x5c;SBox[168] = 0xc2;SBox[169] = 0xd3;SBox[170] = 0xac;SBox[171] = 0x62;SBox[172] = 0x91;SBox[173] = 0x95;SBox[174] = 0xe4;SBox[175] = 0x79;
SBox[176] = 0xe7;SBox[177] = 0xc8;SBox[178] = 0x37;SBox[179] = 0x6d;SBox[180] = 0x8d;SBox[181] = 0xd5;SBox[182] = 0x4e;SBox[183] = 0xa9;SBox[184] = 0x6c;SBox[185] = 0x56;SBox[186] = 0xf4;SBox[187] = 0xea;SBox[188] = 0x65;SBox[189] = 0x7a;SBox[190] = 0xae;SBox[191] = 0x8;
SBox[192] = 0xba;SBox[193] = 0x78;SBox[194] = 0x25;SBox[195] = 0x2e;SBox[196] = 0x1c;SBox[197] = 0xa6;SBox[198] = 0xb4;SBox[199] = 0xc6;SBox[200] = 0xe8;SBox[201] = 0xdd;SBox[202] = 0x74;SBox[203] = 0x1f;SBox[204] = 0x4b;SBox[205] = 0xbd;SBox[206] = 0x8b;SBox[207] = 0x8a;
SBox[208] = 0x70;SBox[209] = 0x3e;SBox[210] = 0xb5;SBox[211] = 0x66;SBox[212] = 0x48;SBox[213] = 0x3;SBox[214] = 0xf6;SBox[215] = 0xe;SBox[216] = 0x61;SBox[217] = 0x35;SBox[218] = 0x57;SBox[219] = 0xb9;SBox[220] = 0x86;SBox[221] = 0xc1;SBox[222] = 0x1d;SBox[223] = 0x9e;
SBox[224] = 0xe1;SBox[225] = 0xf8;SBox[226] = 0x98;SBox[227] = 0x11;SBox[228] = 0x69;SBox[229] = 0xd9;SBox[230] = 0x8e;SBox[231] = 0x94;SBox[232] = 0x9b;SBox[233] = 0x1e;SBox[234] = 0x87;SBox[235] = 0xe9;SBox[236] = 0xce;SBox[237] = 0x55;SBox[238] = 0x28;SBox[239] = 0xdf;
SBox[240] = 0x8c;SBox[241] = 0xa1;SBox[242] = 0x89;SBox[243] = 0xd;SBox[244] = 0xbf;SBox[245] = 0xe6;SBox[246] = 0x42;SBox[247] = 0x68;SBox[248] = 0x41;SBox[249] = 0x99;SBox[250] = 0x2d;SBox[251] = 0xf;SBox[252] = 0xb0;SBox[253] = 0x54;SBox[254] = 0xbb; SBox[255] = 0x16;
   
    int i;
    for(i = 0; i < 256; i++){
        SBox_Inv[SBox[i]] = i;
    }
    for(i = 0; i < 16; i++)
        AES_ShiftRow_Inv[AES_ShiftRow[i]] = i;
    for(i = 0; i < 128; i++) {
        AES_xtime[i] = i << 1;
        AES_xtime[128 + i] = (i << 1) ^ 0x1b;
    }
}

__device__ void AES_Init2(BYTE SBox[], BYTE AES_ShiftRow[], BYTE SBox_Inv[], BYTE AES_xtime[], BYTE AES_ShiftRow_Inv[]) {
    // BYTE AES_ShiftRow[16] ;
    AES_ShiftRow[0]=0;
    AES_ShiftRow[1]=5;
    AES_ShiftRow[2]=10;
    AES_ShiftRow[3]=15;
    AES_ShiftRow[4]=4;
    AES_ShiftRow[5]=9;
    AES_ShiftRow[6]=14;
    AES_ShiftRow[7]=3;
    AES_ShiftRow[8]=8;
    AES_ShiftRow[9]=13;
    AES_ShiftRow[10]=2;
    AES_ShiftRow[11]=7;
    AES_ShiftRow[12]=12;
    AES_ShiftRow[13]=1;
    AES_ShiftRow[14]=6;
    AES_ShiftRow[15]=11;
    

    
SBox_Inv[0] = 0x52;SBox_Inv[1] = 0x9;SBox_Inv[2] = 0x6a;SBox_Inv[3] = 0xd5;SBox_Inv[4] = 0x30;SBox_Inv[5] = 0x36;SBox_Inv[6] = 0xa5;SBox_Inv[7] = 0x38;SBox_Inv[8] = 0xbf;SBox_Inv[9] = 0x40;SBox_Inv[10] = 0xa3;SBox_Inv[11] = 0x9e;SBox_Inv[12] = 0x81;SBox_Inv[13] = 0xf3;SBox_Inv[14] = 0xd7;SBox_Inv[15] = 0xfb;
SBox_Inv[16] = 0x7c;SBox_Inv[17] = 0xe3;SBox_Inv[18] = 0x39;SBox_Inv[19] = 0x82;SBox_Inv[20] = 0x9b;SBox_Inv[21] = 0x2f;SBox_Inv[22] = 0xff;SBox_Inv[23] = 0x87;SBox_Inv[24] = 0x34;SBox_Inv[25] = 0x8e;SBox_Inv[26] = 0x43;SBox_Inv[27] = 0x44;SBox_Inv[28] = 0xc4;SBox_Inv[29] = 0xde;SBox_Inv[30] = 0xe9;SBox_Inv[31] = 0xcb;
SBox_Inv[32] = 0x54;SBox_Inv[33] = 0x7b;SBox_Inv[34] = 0x94;SBox_Inv[35] = 0x32;SBox_Inv[36] = 0xa6;SBox_Inv[37] = 0xc2;SBox_Inv[38] = 0x23;SBox_Inv[39] = 0x3d;SBox_Inv[40] = 0xee;SBox_Inv[41] = 0x4c;SBox_Inv[42] = 0x95;SBox_Inv[43] = 0xb;SBox_Inv[44] = 0x42;SBox_Inv[45] = 0xfa;SBox_Inv[46] = 0xc3;SBox_Inv[47] = 0x4e;
SBox_Inv[48] = 0x8;SBox_Inv[49] = 0x2e;SBox_Inv[50] = 0xa1;SBox_Inv[51] = 0x66;SBox_Inv[52] = 0x28;SBox_Inv[53] = 0xd9;SBox_Inv[54] = 0x24;SBox_Inv[55] = 0xb2;SBox_Inv[56] = 0x76;SBox_Inv[57] = 0x5b;SBox_Inv[58] = 0xa2;SBox_Inv[59] = 0x49;SBox_Inv[60] = 0x6d;SBox_Inv[61] = 0x8b;SBox_Inv[62] = 0xd1;SBox_Inv[63] = 0x25;
SBox_Inv[64] = 0x72;SBox_Inv[65] = 0xf8;SBox_Inv[66] = 0xf6;SBox_Inv[67] = 0x64;SBox_Inv[68] = 0x86;SBox_Inv[69] = 0x68;SBox_Inv[70] = 0x98;SBox_Inv[71] = 0x16;SBox_Inv[72] = 0xd4;SBox_Inv[73] = 0xa4;SBox_Inv[74] = 0x5c;SBox_Inv[75] = 0xcc;SBox_Inv[76] = 0x5d;SBox_Inv[77] = 0x65;SBox_Inv[78] = 0xb6;SBox_Inv[79] = 0x92;
SBox_Inv[80] = 0x6c;SBox_Inv[81] = 0x70;SBox_Inv[82] = 0x48;SBox_Inv[83] = 0x50;SBox_Inv[84] = 0xfd;SBox_Inv[85] = 0xed;SBox_Inv[86] = 0xb9;SBox_Inv[87] = 0xda;SBox_Inv[88] = 0x5e;SBox_Inv[89] = 0x15;SBox_Inv[90] = 0x46;SBox_Inv[91] = 0x57;SBox_Inv[92] = 0xa7;SBox_Inv[93] = 0x8d;SBox_Inv[94] = 0x9d;SBox_Inv[95] = 0x84;
SBox_Inv[96] = 0x90;SBox_Inv[97] = 0xd8;SBox_Inv[98] = 0xab;SBox_Inv[99] = 0x0;SBox_Inv[100] = 0x8c;SBox_Inv[101] = 0xbc;SBox_Inv[102] = 0xd3;SBox_Inv[103] = 0xa;SBox_Inv[104] = 0xf7;SBox_Inv[105] = 0xe4;SBox_Inv[106] = 0x58;SBox_Inv[107] = 0x5;SBox_Inv[108] = 0xb8;SBox_Inv[109] = 0xb3;SBox_Inv[110] = 0x45;SBox_Inv[111] = 0x6;
SBox_Inv[112] = 0xd0;SBox_Inv[113] = 0x2c;SBox_Inv[114] = 0x1e;SBox_Inv[115] = 0x8f;SBox_Inv[116] = 0xca;SBox_Inv[117] = 0x3f;SBox_Inv[118] = 0xf;SBox_Inv[119] = 0x2;SBox_Inv[120] = 0xc1;SBox_Inv[121] = 0xaf;SBox_Inv[122] = 0xbd;SBox_Inv[123] = 0x3;SBox_Inv[124] = 0x1;SBox_Inv[125] = 0x13;SBox_Inv[126] = 0x8a;SBox_Inv[127] = 0x6b;
SBox_Inv[128] = 0x3a;SBox_Inv[129] = 0x91;SBox_Inv[130] = 0x11;SBox_Inv[131] = 0x41;SBox_Inv[132] = 0x4f;SBox_Inv[133] = 0x67;SBox_Inv[134] = 0xdc;SBox_Inv[135] = 0xea;SBox_Inv[136] = 0x97;SBox_Inv[137] = 0xf2;SBox_Inv[138] = 0xcf;SBox_Inv[139] = 0xce;SBox_Inv[140] = 0xf0;SBox_Inv[141] = 0xb4;SBox_Inv[142] = 0xe6;SBox_Inv[143] = 0x73;
SBox_Inv[144] = 0x96;SBox_Inv[145] = 0xac;SBox_Inv[146] = 0x74;SBox_Inv[147] = 0x22;SBox_Inv[148] = 0xe7;SBox_Inv[149] = 0xad;SBox_Inv[150] = 0x35;SBox_Inv[151] = 0x85;SBox_Inv[152] = 0xe2;SBox_Inv[153] = 0xf9;SBox_Inv[154] = 0x37;SBox_Inv[155] = 0xe8;SBox_Inv[156] = 0x1c;SBox_Inv[157] = 0x75;SBox_Inv[158] = 0xdf;SBox_Inv[159] = 0x6e;
SBox_Inv[160] = 0x47;SBox_Inv[161] = 0xf1;SBox_Inv[162] = 0x1a;SBox_Inv[163] = 0x71;SBox_Inv[164] = 0x1d;SBox_Inv[165] = 0x29;SBox_Inv[166] = 0xc5;SBox_Inv[167] = 0x89;SBox_Inv[168] = 0x6f;SBox_Inv[169] = 0xb7;SBox_Inv[170] = 0x62;SBox_Inv[171] = 0xe;SBox_Inv[172] = 0xaa;SBox_Inv[173] = 0x18;SBox_Inv[174] = 0xbe;SBox_Inv[175] = 0x1b;
SBox_Inv[176] = 0xfc;SBox_Inv[177] = 0x56;SBox_Inv[178] = 0x3e;SBox_Inv[179] = 0x4b;SBox_Inv[180] = 0xc6;SBox_Inv[181] = 0xd2;SBox_Inv[182] = 0x79;SBox_Inv[183] = 0x20;SBox_Inv[184] = 0x9a;SBox_Inv[185] = 0xdb;SBox_Inv[186] = 0xc0;SBox_Inv[187] = 0xfe;SBox_Inv[188] = 0x78;SBox_Inv[189] = 0xcd;SBox_Inv[190] = 0x5a;SBox_Inv[191] = 0xf4;
SBox_Inv[192] = 0x1f;SBox_Inv[193] = 0xdd;SBox_Inv[194] = 0xa8;SBox_Inv[195] = 0x33;SBox_Inv[196] = 0x88;SBox_Inv[197] = 0x7;SBox_Inv[198] = 0xc7;SBox_Inv[199] = 0x31;SBox_Inv[200] = 0xb1;SBox_Inv[201] = 0x12;SBox_Inv[202] = 0x10;SBox_Inv[203] = 0x59;SBox_Inv[204] = 0x27;SBox_Inv[205] = 0x80;SBox_Inv[206] = 0xec;SBox_Inv[207] = 0x5f;
SBox_Inv[208] = 0x60;SBox_Inv[209] = 0x51;SBox_Inv[210] = 0x7f;SBox_Inv[211] = 0xa9;SBox_Inv[212] = 0x19;SBox_Inv[213] = 0xb5;SBox_Inv[214] = 0x4a;SBox_Inv[215] = 0xd;SBox_Inv[216] = 0x2d;SBox_Inv[217] = 0xe5;SBox_Inv[218] = 0x7a;SBox_Inv[219] = 0x9f;SBox_Inv[220] = 0x93;SBox_Inv[221] = 0xc9;SBox_Inv[222] = 0x9c;SBox_Inv[223] = 0xef;
SBox_Inv[224] = 0xa0;SBox_Inv[225] = 0xe0;SBox_Inv[226] = 0x3b;SBox_Inv[227] = 0x4d;SBox_Inv[228] = 0xae;SBox_Inv[229] = 0x2a;SBox_Inv[230] = 0xf5;SBox_Inv[231] = 0xb0;SBox_Inv[232] = 0xc8;SBox_Inv[233] = 0xeb;SBox_Inv[234] = 0xbb;SBox_Inv[235] = 0x3c;SBox_Inv[236] = 0x83;SBox_Inv[237] = 0x53;SBox_Inv[238] = 0x99;SBox_Inv[239] = 0x61;
SBox_Inv[240] = 0x17;SBox_Inv[241] = 0x2b;SBox_Inv[242] = 0x4;SBox_Inv[243] = 0x7e;SBox_Inv[244] = 0xba;SBox_Inv[245] = 0x77;SBox_Inv[246] = 0xd6;SBox_Inv[247] = 0x26;SBox_Inv[248] = 0xe1;SBox_Inv[249] = 0x69;SBox_Inv[250] = 0x14;SBox_Inv[251] = 0x63;SBox_Inv[252] = 0x55;SBox_Inv[253] = 0x21;SBox_Inv[254] = 0xc;SBox_Inv[255] = 0x7d;

    int i;
    for(i = 0; i < 16; i++)
        AES_ShiftRow_Inv[AES_ShiftRow[i]] = i;
    for(i = 0; i < 128; i++) {
        AES_xtime[i] = i << 1;
        AES_xtime[128 + i] = (i << 1) ^ 0x1b;
    }
}
 
// Call this function after the last encryption/decryption operation.
void AES_Done() {}


int AES_ExpandKey(BYTE key[], int keyLen) {
    int kl = keyLen, ks, Rcon = 1, i, j;
    BYTE temp[4], temp2[4];
    switch (kl) {
        case 16: ks = 16 * (10 + 1); break;
        case 24: ks = 16 * (12 + 1); break;
        case 32: ks = 16 * (14 + 1); break;
        default: 
        printf("AES_ExpandKey: Only key lengths of 16, 24 or 32 bytes allowed!");
}
    for(i = kl; i < ks; i += 4) {
        memcpy(temp, &key[i-4], 4);
    if (i % kl == 0) {
        temp2[0] = SBox[temp[1]] ^ Rcon;
        temp2[1] = SBox[temp[2]];
        temp2[2] = SBox[temp[3]];
        temp2[3] = SBox[temp[0]];
        memcpy(temp, temp2, 4);
        if ((Rcon <<= 1) >= 256)
            Rcon ^= 0x11b;
}
    else if ((kl > 24) && (i % kl == 16)) {
        temp2[0] = SBox[temp[0]];
        temp2[1] = SBox[temp[1]];
        temp2[2] = SBox[temp[2]];
        temp2[3] = SBox[temp[3]];
        memcpy(temp, temp2, 4);
    }
    for(j = 0; j < 4; j++)
        key[i + j] = key[i + j - kl] ^ temp[j];
    }
    return ks;
}

__global__ void AES_Encrypt(aes_block aes_block_array[], BYTE key[], int keyLen, int block_number) {
    int global_thread_index = blockDim.x*blockIdx.x + threadIdx.x;
    
    __shared__ BYTE AES_ShiftRow[16];
    __shared__ BYTE SBox[256];
    __shared__ BYTE AES_ShiftRow_Inv[16];
    __shared__ BYTE SBox_Inv[256];
    __shared__ BYTE AES_xtime[256];

    if(global_thread_index < block_number){

        if(threadIdx.x == 0 ){
            printf("hello from thread 0\n");
            AES_Init(SBox, AES_ShiftRow, SBox_Inv, AES_xtime, AES_ShiftRow_Inv);
        }
        __syncthreads();
        BYTE block[16]; 

        for(int i=0; i<16; i++){
            block[i] = aes_block_array[global_thread_index].block[i];
        }
        int l = keyLen, i;
        AES_AddRoundKey(block, &key[0]);
        for(i = 16; i < l - 16; i += 16) {
            AES_SubBytes(block, SBox);
           AES_ShiftRows(block, AES_ShiftRow);
           AES_MixColumns(block, AES_xtime);
            AES_AddRoundKey(block, &key[i]);
        }
        AES_SubBytes(block, SBox);
        AES_ShiftRows(block, AES_ShiftRow);
        AES_AddRoundKey(block, &key[i]);

        for(int i=0; i<16; i++){
  
         aes_block_array[global_thread_index].block[i] = block[i];
        }
        
    }
}


__global__ void AES_Decrypt(aes_block aes_block_array[], BYTE key[], int keyLen, int block_number) {
    int global_thread_index = blockDim.x*blockIdx.x + threadIdx.x;
    __shared__ BYTE AES_ShiftRow[16];
    __shared__ BYTE SBox[256];
    __shared__ BYTE AES_ShiftRow_Inv[16];
    __shared__ BYTE SBox_Inv[256];
    __shared__ BYTE AES_xtime[256];

    if(global_thread_index < block_number){

        if(threadIdx.x == 0 ){

            AES_Init2(SBox, AES_ShiftRow, SBox_Inv, AES_xtime, AES_ShiftRow_Inv);
        }
        __syncthreads();
        BYTE block[16]; 
        for(int i=0; i<16; i++){
            block[i] = aes_block_array[global_thread_index].block[i];

        }
int l = keyLen, i;
AES_AddRoundKey(block, &key[l - 16]);
AES_ShiftRows(block, AES_ShiftRow_Inv);
AES_SubBytes(block, SBox_Inv);
for(i = l - 32; i >= 16; i -= 16) {
    AES_AddRoundKey(block, &key[i]);
    AES_MixColumns_Inv(block, AES_xtime);
    AES_ShiftRows(block, AES_ShiftRow_Inv);
AES_SubBytes(block, SBox_Inv);
}
AES_AddRoundKey(block, &key[0]);
        for(int i=0; i<16; i++){

         aes_block_array[global_thread_index].block[i] = block[i];
        }
}
}


// testing
int main(int argc, char* argv[]) {


    ifstream ifs;
    ifs.open(argv[1], std::ifstream::binary);
    if(!ifs){
        cerr<<"Cannot open file"<<endl;
        exit(1);
    }
    ifs.seekg(0, ios::end);
    int infileLength = ifs.tellg();
    ifs.seekg (0, ios::beg);
    cout<<"Length of input file: "<<infileLength<<endl;


int block_number = infileLength/16 ;
int number_of_zero_pending = infileLength%16;
aes_block* aes_block_array;

BYTE key[16 * (14 + 1)];
int keyLen = 0;
,
int blockLen = 16;

ifstream key_fp;
key_fp.open(argv[2]);
while(key_fp.peek()!=EOF)
{
        key_fp>>key[keyLen];
        if(key_fp.eof())
            break;
        keyLen++;
}

cout<<keyLen<<endl;
switch (keyLen)
{
  case 16:break;
  case 24:break;
  case 32:break;
  default:printf("Key length should be 128, 192, 256bits\n"); return 0;
}

int expandKeyLen = AES_ExpandKey(key, keyLen);


if(number_of_zero_pending != 0)
    aes_block_array = new aes_block [ block_number + 1];
else
    aes_block_array = new aes_block[ block_number ];
char temp[16];

FILE* en_fp;
FILE* de_fp;

en_fp = fopen(argv[3], "wb");
de_fp = fopen(argv[4], "wb");
for(int i=0; i<block_number; i++){
    
    ifs.read(temp, 16);
    for(int j=0; j<16; j++){
        aes_block_array[i].block[j] = (unsigned char)temp[j];
    }
}
if(number_of_zero_pending != 0)
{
    ifs.read(temp, number_of_zero_pending);
    for(int j=0; j<16; j++){
        aes_block_array[block_number].block[j] = (unsigned char)temp[j];
    }
    for(int j=1; j<=16-number_of_zero_pending; j++)
        aes_block_array[block_number].block[16-j] = '\0';
    block_number++;
}


hipSetDevice(0);	
hipDeviceProp_t prop;
hipGetDeviceProperties(&prop, 0);
int num_sm = prop.multiProcessorCount; 

aes_block *cuda_aes_block_array;
BYTE *cuda_key;


int thrdperblock = block_number/num_sm;
if(block_number%num_sm>0)
    thrdperblock++;

if(thrdperblock>1024){
    thrdperblock = 1024;
    num_sm = block_number/1024;
    if(block_number%1024>0){
        num_sm++;
    }
}
dim3 ThreadperBlock(thrdperblock);

printf("\nThreads per block: %d\n", thrdperblock);

dim3 BlockperGrid(num_sm);
hipMalloc(&cuda_aes_block_array, block_number*sizeof(class aes_block));
hipMalloc(&cuda_key,16*15*sizeof(BYTE) );
hipMemcpy(cuda_aes_block_array, aes_block_array, block_number*sizeof(class aes_block), hipMemcpyHostToDevice);
hipMemcpy(cuda_key, key, 16*15*sizeof(BYTE), hipMemcpyHostToDevice);


AES_Encrypt <<< BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_key, expandKeyLen, block_number);

hipMemcpy(aes_block_array, cuda_aes_block_array, block_number*sizeof(class aes_block), hipMemcpyDeviceToHost);

for(int i=0; i<block_number-1; i++){
    fileOnePrintBytes(aes_block_array[i].block, blockLen, en_fp);
}
if(number_of_zero_pending == 0)
    fileOnePrintBytes(aes_block_array[block_number-1].block, blockLen, en_fp);
else 
    fileOnePrintBytes(aes_block_array[block_number-1].block, blockLen, en_fp);


AES_Decrypt <<< BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_key, expandKeyLen, block_number);

hipMemcpy(aes_block_array, cuda_aes_block_array, block_number*sizeof(class aes_block), hipMemcpyDeviceToHost);

for(int i=0; i<block_number-1; i++){
    fileTwoPrintBytes(aes_block_array[i].block, blockLen, de_fp);
}
if(number_of_zero_pending == 0)
    fileTwoPrintBytes(aes_block_array[block_number-1].block, blockLen, de_fp);
else 
    fileThreePrintBytes(aes_block_array[block_number-1].block, blockLen, de_fp);


AES_Done();
fclose(en_fp);
fclose(de_fp);

    return 0;
}
